#include "hip/hip_runtime.h"
// OptiX Implementation for NEBULA
// OptiXRayTracing.cu

#include <optix.h>
#include <hip/hip_runtime.h>

// OptiX ray generation program
extern "C" __global__ void __raygen__nebula()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    
    // Camera setup
    const float3 eye = params.eye;
    const float3 U = params.U;
    const float3 V = params.V;
    const float3 W = params.W;
    
    // Generate ray
    const float2 d = 2.0f * make_float2(
        static_cast<float>(idx.x) / static_cast<float>(dim.x),
        static_cast<float>(idx.y) / static_cast<float>(dim.y)
    ) - 1.0f;
    
    const float3 direction = normalize(d.x * U + d.y * V + W);
    
    // Trace ray through neural galaxy
    unsigned int p0, p1, p2;
    p0 = __float_as_uint(1.0f);  // Current luminosity
    p1 = __float_as_uint(0.0f);  // Accumulated spectrum R
    p2 = __float_as_uint(0.0f);  // Accumulated spectrum G
    
    optixTrace(
        params.handle,
        eye,
        direction,
        0.01f,                // tmin
        1e16f,                // tmax
        0.0f,                 // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        0,                    // SBT offset
        0,                    // SBT stride
        0,                    // missSBTIndex
        p0, p1, p2
    );
    
    // Write result
    params.frame_buffer[idx.y * dim.x + idx.x] = make_float4(
        __uint_as_float(p0),
        __uint_as_float(p1),
        __uint_as_float(p2),
        1.0f
    );
}

// Sphere intersection for neurons
extern "C" __global__ void __intersection__sphere()
{
    const float3 ray_orig = optixGetObjectRayOrigin();
    const float3 ray_dir = optixGetObjectRayDirection();
    const float ray_tmin = optixGetRayTmin();
    const float ray_tmax = optixGetRayTmax();
    
    const unsigned int primIdx = optixGetPrimitiveIndex();
    const float3 center = params.neurons[primIdx].position;
    const float radius = params.neurons[primIdx].radius;
    
    const float3 O = ray_orig - center;
    const float b = dot(O, ray_dir);
    const float c = dot(O, O) - radius * radius;
    const float discriminant = b * b - c;
    
    if (discriminant > 0.0f)
    {
        const float sqrtd = sqrtf(discriminant);
        const float t0 = -b - sqrtd;
        const float t1 = -b + sqrtd;
        
        if (t0 >= ray_tmin && t0 <= ray_tmax)
        {
            optixReportIntersection(t0, 0);
        }
        else if (t1 >= ray_tmin && t1 <= ray_tmax)
        {
            optixReportIntersection(t1, 0);
        }
    }
}

// Closest hit program for neurons
extern "C" __global__ void __closesthit__neuron()
{
    const unsigned int primIdx = optixGetPrimitiveIndex();
    
    // Get neuron data
    NeuronData& neuron = params.neurons[primIdx];
    
    // Calculate illumination
    float3 hitPoint = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    float3 normal = normalize(hitPoint - neuron.position);
    
    // Accumulate neuron's light contribution
    float luminosity = neuron.luminosity;
    float3 spectrum = make_float3(
        neuron.spectrum.x,
        neuron.spectrum.y,
        neuron.spectrum.z
    );
    
    // Pass accumulated light
    unsigned int p0 = optixGetPayload_0();
    unsigned int p1 = optixGetPayload_1();
    unsigned int p2 = optixGetPayload_2();
    
    float current_lum = __uint_as_float(p0);
    float current_r = __uint_as_float(p1);
    float current_g = __uint_as_float(p2);
    
    // Add this neuron's contribution
    current_lum += luminosity * neuron.activation;
    current_r += spectrum.x * luminosity;
    current_g += spectrum.y * luminosity;
    
    optixSetPayload_0(__float_as_uint(current_lum));
    optixSetPayload_1(__float_as_uint(current_r));
    optixSetPayload_2(__float_as_uint(current_g));
    
    // Continue tracing (for volumetric effect)
    if (current_lum < 10.0f)  // Max accumulation
    {
        float3 newDirection = optixGetWorldRayDirection();
        
        // Scatter direction (simplified)
        newDirection = normalize(newDirection + normal * 0.1f);
        
        optixTrace(
            params.handle,
            hitPoint + normal * 0.01f,
            newDirection,
            0.01f,
            1e16f,
            0.0f,
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_NONE,
            0, 0, 0,
            p0, p1, p2
        );
    }
}